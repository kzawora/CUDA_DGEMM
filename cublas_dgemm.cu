#include "hip/hip_runtime.h"
#include "common_header.h"

cudaReturnValue cublasDgemmWrapper(
    hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k,
    const double* alpha,
    const double* A, int lda,
    const double* B, int ldb,
    const double* beta,
    double* C, int ldc) {

    double* dev_A = 0;
    const int dev_A_size = m * k * sizeof(double);
    double* dev_B = 0;
    const int dev_B_size = n * k * sizeof(double);
    double* dev_C = 0;
    const int dev_C_size = m * n * sizeof(double);

    hipError_t cudaStatus;
    double executionTime = -1.;
    hipblasHandle_t handle;
    hipblasStatus_t stat;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)& dev_A, dev_A_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)& dev_B, dev_B_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)& dev_C, dev_C_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "CUBLAS initialization failed\n");
        cudaStatus = hipErrorNotSupported;
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_A, A, dev_A_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_B, B, dev_B_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_C, C, dev_C_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // start time measurement
    clock_t t;
    t = clock();

    // execute hipblasDgemm
    stat = hipblasDgemm(
        handle,
        transa, transb,
        m, n, k,
        alpha,
        dev_A, lda,
        dev_B, ldb,
        beta,
        dev_C, ldc
    );

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipblasDgemm launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // measure time
    t = clock() - t;
    executionTime = ((double)t) / CLOCKS_PER_SEC;

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(C, dev_C, dev_C_size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    return { cudaStatus, executionTime };
}