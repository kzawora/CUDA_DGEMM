#include "hip/hip_runtime.h"
#include ""
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

hipError_t cublasDgemmWrapper(
    hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k,
    const double* alpha,
    const double* A, int lda,
    const double* B, int ldb,
    const double* beta,
    double* C, int ldc);

hipError_t myDgemmHostCode(
    hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k,
    const double* alpha,
    const double* A, int lda,
    const double* B, int ldb,
    const double* beta,
    double* C, int ldc);

__global__ void myDgemmKernel_naive(
    hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k,
    const double alpha,
    const double* A, int lda,
    const double* B, int ldb,
    const double beta,
    double* C, int ldc)
{
    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;
    int mat_c_idx = tid_x + tid_y * ldc;
    C[mat_c_idx] *= beta;
    for (int i = 0; i < k; i++) {
        int mat_a_idx = transa == HIPBLAS_OP_T ? tid_x * lda + i : tid_x + i * lda;
        int mat_b_idx = transb == HIPBLAS_OP_T ? tid_y + i * ldb : tid_y * ldb + i;
        C[mat_c_idx] += A[mat_a_idx] * B[mat_b_idx];
    }
}

void printMatrixColMajor(const double* matrix, int width, int height) {
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < height; j++) {
            printf("%lf ", matrix[i + j * width]);
        }
        printf("\n");
    }
}

int main() {
    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_T;
    const int m = 2;
    const int n = 4;
    const int k = 3;
    const double alpha = 1.;
    const double beta = 2.;
    const int lda = transa == HIPBLAS_OP_N ? m : k;
    const int ldb = transb == HIPBLAS_OP_N ? k : n;
    const int ldc = m;

    const double A[m * k] = {
        1.,2.,3.,
        4.,5.,6.
    };

    const double B[n * k] = {
        1.,2.,3., 4.,
        5.,6.,7.,8.,
        9.,10.,11.,12.
    };

    double C_myKernel[m * n] = {
        1.,1.,1., 1.,
        1.,1.,1., 1.
    };
    double C_cublas[m * n] = {
        1.,1.,1., 1.,
        1.,1.,1., 1.
    };

    // run cublasGemm
    hipError_t hipblasStatus_t = cublasDgemmWrapper(transa, transb, m, n, k, &alpha, A, lda, B, ldb, &beta, C_cublas, ldc);
    if (hipblasStatus_t != hipSuccess) {
        fprintf(stderr, "cuBlasDgemm failed!");
        return 1;
    }


    // run myKernel
    hipError_t myKernelStatus = myDgemmHostCode(transa, transb, m, n, k, &alpha, A, lda, B, ldb, &beta, C_myKernel, ldc);
    if (myKernelStatus != hipSuccess) {
        fprintf(stderr, "myKernel failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    // print results
    printf("hipblasDgemm:\n");
    printMatrixColMajor(C_cublas, ldc, n);
    printf("myKernel:\n");
    printMatrixColMajor(C_myKernel, ldc, n);

    // compare results
    int errorCounter = 0;
    double epsilon = 1e-9;
    for (int i = 0; i < ldc; i++) {
        for (int j = 0; j < n; j++) {
            if (fabs(C_myKernel[i + j * ldc] - C_cublas[i + j * ldc]) > epsilon) {
                errorCounter++;
                printf("Value mismatch at (%d,%d):\n  Expected: %lf\n  Actual: %lf\n", i, j, C_cublas[i + j * ldc], C_myKernel[i + j * ldc]);
            }
        }
    }
    if (errorCounter == 0)
        printf("No mismatches found.\n");
    else
        printf("%d mismatch(es) found.\n", errorCounter);

    return 0;
}


hipError_t myDgemmHostCode(
    hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k,
    const double* alpha,
    const double* A, int lda,
    const double* B, int ldb,
    const double* beta,
    double* C, int ldc) {

    double* dev_A = 0;
    const int dev_A_size = m * k * sizeof(double);
    double* dev_B = 0;
    const int dev_B_size = n * k * sizeof(double);
    double* dev_C = 0;
    const int dev_C_size = m * n * sizeof(double);

    hipError_t cudaStatus;
    hipblasHandle_t handle;
    hipblasStatus_t stat;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)& dev_A, dev_A_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)& dev_B, dev_B_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)& dev_C, dev_C_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "CUBLAS initialization failed\n");
        cudaStatus = hipErrorNotSupported;
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_A, A, dev_A_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_B, B, dev_B_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_C, C, dev_C_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    dim3 threadsPerBlock(m, n);
    dim3 numBlocks(1);
    // Launch a kernel on the GPU with one thread for each element.
    myDgemmKernel_naive<<<numBlocks, threadsPerBlock>>> (
        transa, transb,
        m, n, k,
        *alpha,
        dev_A, lda,
        dev_B, ldb,
        *beta,
        dev_C, ldc
        );

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(C, dev_C, dev_C_size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    return cudaStatus;
}


hipError_t cublasDgemmWrapper(
    hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k,
    const double* alpha,
    const double* A, int lda,
    const double* B, int ldb,
    const double* beta,
    double* C, int ldc) {

    double* dev_A = 0;
    const int dev_A_size = m * k * sizeof(double);
    double* dev_B = 0;
    const int dev_B_size = n * k * sizeof(double);
    double* dev_C = 0;
    const int dev_C_size = m * n * sizeof(double);

    hipError_t cudaStatus;
    hipblasHandle_t handle;
    hipblasStatus_t stat;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)& dev_A, dev_A_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)& dev_B, dev_B_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)& dev_C, dev_C_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "CUBLAS initialization failed\n");
        cudaStatus = hipErrorNotSupported;
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_A, A, dev_A_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_B, B, dev_B_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_C, C, dev_C_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    stat = hipblasDgemm(
        handle,
        transa, transb,
        m, n, k,
        alpha,
        dev_A, lda,
        dev_B, ldb,
        beta,
        dev_C, ldc
    );

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipblasDgemm launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(C, dev_C, dev_C_size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    return cudaStatus;
}
